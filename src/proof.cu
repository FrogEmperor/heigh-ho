#include "hip/hip_runtime.h"
#include "proof.cuh"
#include "sha256_gpu.cuh"

__device__ int stringSize(char* string){
    int size = 0;
    while(string[size] != 0)
        size++;
    return size;
}

void swap(char *x, char *y) {
    char t = *x; *x = *y; *y = t;
}
 
char* reverse(char *buffer, int i, int j)
{
    while (i < j) {
        swap(&buffer[i++], &buffer[j--]);
    }
 
    return buffer;
}

__device__ int digitCount(int n){
    int count = 0;
    
    do {
        n /= 10;
        ++count;
    } while (n != 0);
    
    return count;
}

__device__ char* itoa(int value){
    char* buffer = (char*)malloc(digitCount(value) * sizeof(char));
    int base = 10;
 
    int n = abs(value);
 
    int i = 0;
    while (n)
    {
        int r = n % base;
 
        if (r >= 10) {
            buffer[i++] = 65 + (r - 10);
        }
        else {
            buffer[i++] = 48 + r;
        }
 
        n = n / base;
    }
 
    if (i == 0) {
        buffer[i++] = '0';
    }
 
    if (value < 0 && base == 10) {
        buffer[i++] = '-';
    }
 
    buffer[i] = '\0'; // null terminate string
 
    return reverse(buffer, 0, i - 1);
}

__device__ char* joinStrings(char* first, char* second){
    int firstSize = stringSize(first); 
    int secondSize = stringSize(second);
    int totalSize = firstSize + secondSize;
    char* result = (char*)malloc((totalSize + 1) * sizeof(char));
    for(int i = 0; i < firstSize; i++)
        result[i] = first[i];
    for(int i = 0; i < secondSize; i++)
        result[firstSize + i] = second[i];
    result[totalSize] = 0;
    return result;
}

__device__ char* substring(char* string, int len){
    char* result = (char*)malloc((len+1) * sizeof(char));
    for(int i = 0; i < len; i++)
        result[i] = string[i];
    result[len] = 0;
    return result;
}

__device__ char* repeatChar(char c, int len){
    char* result = (char*)malloc((len+1)*sizeof(char));
    for(int i = 0; i < len; i++)
        result[i] = c;
    result[len] = 0;
    return result;
}

__device__ int compareStrings(char* first, char* second, int len){
    for(int i = 0; i < len; i++)
        if(first[i] != second[i])
            return 0;
    return 1;
}

__device__ int validProof(int lastProof, int currentProof, int level){
    char* lastProofChar = itoa(lastProof);
    char* currentProofChar = itoa(currentProof);
    char* guess = joinStrings(lastProofChar, currentProofChar);
    free(lastProofChar); 
    free(currentProofChar);
    char guessHash[65] = {0};
    sha256_easy_hash_hex(guess, stringSize(guess), guessHash);
    free(guess);
    char* sub = substring(guessHash, level);
    char* zero = repeatChar('0', level);
    int result = compareStrings(sub, zero, level);
    free(sub);
    free(zero);
    return result;
}

__device__ int proofOfWork(int lastProof, int level){
    int proof = 0;
    while(!validProof(lastProof, proof, level)) {
        proof += 1;
    }
    return proof;
}