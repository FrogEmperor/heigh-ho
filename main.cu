#include "hip/hip_runtime.h"
#include <iostream>
#include "blockchain.hpp"
#include "hip/hip_runtime.h"

__global__ void kernel(){

}

int main() {
    Blockchain btc = Blockchain(7);
    btc.newTransaction("oscar", "manolo", 1);
    btc.newTransaction("manolo", "dagz", 2);

    int lastProof = btc.lastBlock().proof;
    int newProof = btc.proofOfWork(lastProof);
    btc.newBlock(newProof, std::to_string(lastProof));
    kernel<<<1,1>>>();
    return 0;
}
