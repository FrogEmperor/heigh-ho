#include "hip/hip_runtime.h"
#include "sha256_gpu.cuh"
#include "proof.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(long long int* buffer, long long int lastProof, long long int currentProof, long long int level){
    long long int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(validProof(lastProof, i+currentProof, level)==1)
        *buffer = i+currentProof;
}

__host__ void callKernel(long long int* d_buffer, long long int* h_buffer, long long int lastProof, long long int level){
    long long int count = 0;
    long long int t = 128;
    long long int b = 4097 / t + 1;
    printf("%lld\n", *h_buffer);
    while(*h_buffer == 0) 
    {
        kernel<<<b,t>>>(d_buffer, lastProof, count * t * b, level);
        hipMemcpy(h_buffer, d_buffer, sizeof(int*), hipMemcpyDeviceToHost);
        count++;
        if(count % 300 == 0)
            printf("%lld\n", count);
    }
    printf("%lld\n", count);
}

int main(){
    long long int lastProof= 1324;
    long long int level = 6;
    long long int* h_buffer = (long long int*)malloc(sizeof(long long int*));
    long long int* d_buffer;
    *h_buffer=0;
    hipMalloc((void**)&d_buffer, sizeof(long long int*));
    callKernel(d_buffer, h_buffer, lastProof, level);

    printf("%lld",*h_buffer);
}