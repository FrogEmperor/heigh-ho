#include "hip/hip_runtime.h"
#include "sha256_gpu.cuh"
#include "proof.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(int* buffer){
    *buffer = proofOfWork(12345,2);
}


int main(){
    int h_buffer[1];
    int* d_buffer;
    hipMalloc((void**)&d_buffer, sizeof(int));
    kernel<<<1,1>>>(d_buffer);
    hipMemcpy(h_buffer, d_buffer, sizeof(int), hipMemcpyDeviceToHost);
    printf("%d",*h_buffer);
}