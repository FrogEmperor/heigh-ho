#include "hip/hip_runtime.h"
#include "sha256_gpu.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(char* buffer){
    char* string = {"abc"};
    int count = 0;
    while(string[count] != 0)
        count++;
    sha256_easy_hash_hex(string, count,buffer);
}

int main(){
    char* h_buffer[65] = {0};
    char* d_buffer;
    hipMalloc((void**)&d_buffer, sizeof(char) * 65);
    kernel<<<1,1>>>(d_buffer);
    hipMemcpy(h_buffer, d_buffer, sizeof(char)*65, hipMemcpyDeviceToHost);
    printf("%s",h_buffer);
}